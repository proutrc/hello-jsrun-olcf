/**********************************************************

"Hello World"-type program to test different jsrun layouts.

Written by Tom Papatheodore

**********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <mpi.h>
#include <sched.h>
#include <nvml.h>
#include <omp.h>

int main(int argc, char *argv[]){

	MPI_Init(&argc, &argv);

	int size;
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	int rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);

	char name[MPI_MAX_PROCESSOR_NAME];
	int resultlength;
	MPI_Get_processor_name(name, &resultlength);

	hipError_t cuErr;

	// Find how many GPUs CUDA runtime says are available
	int num_devices = 0;
	cuErr = hipGetDeviceCount(&num_devices);

	// Set output based on command line argument
	// => verbose shows BusID and UUID for GPUs
	char output_flag[64];
	strcpy(output_flag, "not_verbose");
	if(argc > 1){ 
		if(strlen(argv[1]) >= sizeof(output_flag)){
			printf("Argument too long: %s\n", argv[1]);
			exit(0);
		}
		else{
			strcpy(output_flag, argv[1]);
		}
	}

	int hwthread;
	int num_threads = 0;
	int thread_id = 0;

	#pragma omp parallel default(shared)
	{
		num_threads = omp_get_num_threads();
	}

	if(rank == 0){
		printf("\n---------- MPI Ranks: %d, OpenMP Threads: %d, GPUs per Resource Set: %d ----------\n", size, num_threads, num_devices);
	}

	if(num_devices == 0){
		#pragma omp parallel default(shared) private(hwthread, thread_id)
		{
			thread_id = omp_get_thread_num();
			hwthread = sched_getcpu();

			printf("MPI Rank %03d of %03d on HWThread %03d of Node %s, OMP_threadID %d of %d\n", rank, size, hwthread, name, thread_id, num_threads);
		}
	}
	else{

		// NVML is needed to query the UUID of GPUs, which
		// allows us to check which GPU is actually being used
		// by each MPI rank
		nvmlReturn_t result;
		result = nvmlInit();
		if(NVML_SUCCESS != result){
			printf("Failed to initialize NVML: %s\n", nvmlErrorString(result));
			exit(0);
		}
		char uuid[NVML_DEVICE_UUID_BUFFER_SIZE];
		char busid[64];
		int gpu_id;

		char uuid_list[1024] = "";
		char busid_list[1024] = "";
		char rt_gpu_id_list[1024] = "";
		char gpu_id_list[1024] = "";

		char c_i[12] = "";
		char c_gpu_id[12] = "";

		// Loop over the GPUs available to each MPI rank
		for(int i=0; i<num_devices; i++){

			cuErr = hipSetDevice(i);
			if(hipSuccess != cuErr){
				printf("CUDA Error - hipSetDevice: %s/n", hipGetErrorString(cuErr));
				exit(0);
			}

			// Get the PCIBusId for each GPU and use it to query for UUID
			cuErr = hipDeviceGetPCIBusId(busid, 64, i);
			if(hipSuccess != cuErr){
				printf("CUDA Error - hipDeviceGetPCIBusId: %s/n", hipGetErrorString(cuErr));
				exit(0);
			}

			// Get UUID for the device based on busid
			nvmlDevice_t device;

			result = nvmlDeviceGetHandleByPciBusId(busid, &device);
			if(NVML_SUCCESS != result){
				printf("nvmlDeviceGetHandleByPciBusId Failed: %s\n", nvmlErrorString(result));
				exit(0);
			}	

			result = nvmlDeviceGetUUID(device, uuid, NVML_DEVICE_UUID_BUFFER_SIZE);
			if(NVML_SUCCESS != result){
				printf("nvmlDeviceGetUUID Failed: %s\n", nvmlErrorString(result));
				exit(0);
			}

			// Map DomainID and BusID to node-local GPU ID
			if(strcmp(busid, "0004:04:00.0") == 0){
				gpu_id = 0;
			}else if(strcmp(busid, "0004:05:00.0") == 0){
				gpu_id = 1;
			}else if(strcmp(busid, "0004:06:00.0") == 0){
				gpu_id = 2;
			}else if(strcmp(busid, "0035:03:00.0") == 0){
				gpu_id = 3;
			}else if(strcmp(busid, "0035:04:00.0") == 0){
				gpu_id = 4;
			}else if(strcmp(busid, "0035:05:00.0") == 0){
				gpu_id = 5;
			}else if(strcmp(busid, "0002:01:00.0") == 0){
				gpu_id = 0;
			}else if(strcmp(busid, "0003:01:00.0") == 0){
				gpu_id = 1;
			}else if(strcmp(busid, "0006:01:00.0") == 0){
				gpu_id = 2;
			}else if(strcmp(busid, "0007:01:00.0") == 0){
				gpu_id = 3;
			}else{
				printf("The BusID (%s) did not map correctly to a GPU. Exiting...\n", busid);
				exit(0);
			}

			// Concatenate per-MPIrank GPU info into strings for printf
			sprintf(c_i, "%d", i);
			sprintf(c_gpu_id, "%d", gpu_id);

			strcat(rt_gpu_id_list, strcat(c_i, " "));
			strcat(gpu_id_list, strcat(c_gpu_id, " "));
			strcat(busid_list, strcat(busid, " "));
			strncat(uuid_list, uuid, 10*sizeof(char));
			strcat(uuid_list, " ");

		}

		#pragma omp parallel default(shared) private(hwthread, thread_id)
		{
			thread_id = omp_get_thread_num();
			hwthread = sched_getcpu();

			if(strcmp(output_flag, "verbose") == 0){
				printf("MPI Rank %03d, OMP_thread %02d on HWThread %03d of Node %s - RT_GPU_id %s: GPU_id %s, BusID %s, UUID %s\n", rank, thread_id, hwthread, name, rt_gpu_id_list, gpu_id_list, busid_list, uuid_list);
			}
			else{
				printf("MPI Rank %03d, OMP_thread %02d on HWThread %03d of Node %s - RT_GPU_id %s: GPU_id %s\n", rank, thread_id, hwthread, name, rt_gpu_id_list, gpu_id_list);	
			}
		}

		result = nvmlShutdown();
		if(NVML_SUCCESS != result){
			printf("NVML Failed to Shutdown: %s\n", nvmlErrorString(result));
			exit(0);
		}

	}

	MPI_Finalize();

	return 0;
}
